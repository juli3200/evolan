#include "hip/hip_runtime.h"
#include "render.h"
#include <iostream>
#include <stdio.h>

__global__ void render_kernel(unsigned char* data, unsigned char* image, unsigned char* colors) {
    const int index = threadIdx.x + threadIdx.y * blockDim.x;

    // multiply the index by 3 because each pixel has 3 values (r, g, b)
    
    image[index* 3] = colors[data[index]*3];
    image[index* 3 + 1] = colors[data[index] * 3 + 1];
    image[index* 3 + 2] = colors[data[index] * 3 + 2];
    

}



/*
param: data - array of data to render in the form of [0,1,1,2,1,0,0,0,...]]
    where each number represents a color in the colors array

param: colors - array of colors to render in the form of [r, g, b, r, g, b, ...] is limited to 16 colors

*/
unsigned char* render(unsigned char* data, unsigned char* colors, int width, int height){

    // check input
    if (width <= 0 || height <= 0) {
        std::cout << "width and height must be greater than 0" << std::endl;
        return NULL;
    }

    // size of data
    int data_size = sizeof(unsigned char) * width * height;

    // creating an array 3 times bigger then the data array
    // because each pixel has 3 values (r, g, b)
    unsigned char* image = (unsigned char*)malloc(sizeof(unsigned char) * width * height * 3);
    
    std::cout << sizeof(data_size) << std::endl;

    // create pointers to the data and colors arrays gpu
    unsigned char* dev_data;
    unsigned char* dev_colors;
    unsigned char* dev_image;

    // allocate memory on the gpu
    hipMalloc((void**)&dev_data, data_size);
    hipMalloc((void**)&dev_colors, sizeof(unsigned char) * 3  * /*colors array is limited to 16 colors*/16);
    hipMalloc((void**)&dev_image, data_size * 3);

    // copy the data and colors arrays to the gpu
    hipMemcpy(dev_data, data, data_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_colors, colors, sizeof(unsigned char) * 3  * 16, hipMemcpyHostToDevice);

    dim3 grid(width, height);

    render_kernel<<<1, grid>>>(dev_data, dev_image, dev_colors);

    hipMemcpy(image, dev_image, data_size*3, hipMemcpyDeviceToHost);

    hipFree(dev_data);
    hipFree(dev_colors);
    hipFree(dev_image);
    
    return image;
}



unsigned char* cpu_render(unsigned char* data, unsigned char* colors, int width, int height) {
    // Check input
    if (width <= 0 || height <= 0) {
        std::cout << "Width and height must be greater than 0" << std::endl;
        return NULL;
    }

    unsigned char* image = (unsigned char*)malloc(sizeof(unsigned char) * width * height * 3);

    // Render on CPU
    for (int j = 0; j < height; ++j) {
        for (int i = 0; i < width; ++i) {
            const int index = j * width + i;
            image[index * 3] = colors[data[index] * 3];
            image[index * 3 + 1] = colors[data[index] * 3 + 1];
            image[index * 3 + 2] = colors[data[index] * 3 + 2];
        }
    }
    return image;
}