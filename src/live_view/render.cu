#include "hip/hip_runtime.h"
#include "render.h"
#include <iostream>

__global__ void render_kernel(int* a, int* b, int* c) {
    
}

void hello() {
    int a[100];
    int b[100];
    int c[100];

    for (int i = 0; i < 100; i++) {
        a[i] = i;
        b[i] = i;
    }

    int* dev_a;
    int* dev_b;
    int* dev_c;

    hipMalloc((void**)&dev_a, 100 * sizeof(int));
    hipMalloc((void**)&dev_b, 100 * sizeof(int));
    hipMalloc((void**)&dev_c, 100 * sizeof(int));

    hipMemcpy(dev_a, a, 100 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, 100 * sizeof(int), hipMemcpyHostToDevice);

    render_kernel<<<1, 100>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, 100 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);


    
    std::cout << "Hello from cuda!" << std::endl;
}

/*
param: data - array of data to render in the form of [0,1,1,2,1,0,0,0,...]]
    where each number represents a color in the colors array

param: colors - array of colors to render in the form of [r, g, b, r, g, b, ...]

*/
unsigned char* render(unsigned char* data, unsigned char* colors, int width, int height){
    // creating an array 3 times bigger then the data array
    // because each pixel has 3 values (r, g, b)
    unsigned char* image = new unsigned char[height * width * 3];
    // initializing the image array to 0
    for (int i = 0; i < 100 * 100 * 3; i++) {
        image[i] = 0;
    }

    
    for (int i = 0; i < 100 * 100; i++) {
        int color = data[i];
        image[i * 3] = colors[color * 3];
        image[i * 3 + 1] = colors[color * 3 + 1];
        image[i * 3 + 2] = colors[color * 3 + 2];
    }
    return image;
}